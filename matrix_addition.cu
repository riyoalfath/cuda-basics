
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CPU function for Matrix Addition
void matrixAddCPU(const float *A, const float *B, float *C, int M, int N) {
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      int idx = i * N + j;
      C[idx] = A[idx] + B[idx];
    }
  }
}

// CUDA Kernel for 2D Matrix Addition
__global__ void matrixAdd(const float *A, const float *B, float *C, int M,
                          int N) {
  // Calculate the global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Boundary check
  if (row < M && col < N) {
    int idx = row * N + col;
    C[idx] = A[idx] + B[idx];
  }
}

int main() {
  // Matrix dimensions
  int M = 1024;
  int N = 1024;
  int numElements = M * N;
  size_t size = numElements * sizeof(float);

  // Host matrices using malloc
  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C_gpu = (float *)malloc(size);
  float *h_C_cpu = (float *)malloc(size);

  // Initialize host matrices
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = (float)rand() / RAND_MAX;
    h_B[i] = (float)rand() / RAND_MAX;
  }

  // --- CPU Execution and Timing ---
  clock_t start_cpu = clock();
  matrixAddCPU(h_A, h_B, h_C_cpu, M, N);
  clock_t end_cpu = clock();
  double cpu_duration_ms =
      1000.0 * (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC;
  printf("CPU execution time: %f ms\n", cpu_duration_ms);

  // --- GPU Execution and Timing ---
  // Device pointers
  float *d_A, *d_B, *d_C;

  // Allocate memory on the device
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  // Copy data from host to device
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Define thread block and grid dimensions
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

  // Use CUDA events for accurate GPU timing
  hipEvent_t start_gpu, stop_gpu;
  hipEventCreate(&start_gpu);
  hipEventCreate(&stop_gpu);

  // Launch the kernel
  hipEventRecord(start_gpu);
  matrixAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N);
  hipEventRecord(stop_gpu);
  hipEventSynchronize(stop_gpu);

  float gpu_duration_ms = 0;
  hipEventElapsedTime(&gpu_duration_ms, start_gpu, stop_gpu);
  printf("GPU kernel execution time: %f ms\n", gpu_duration_ms);

  // Copy the result back from device to host
  hipMemcpy(h_C_gpu, d_C, size, hipMemcpyDeviceToHost);

  // --- Verification ---
  double error = 0.0;
  for (int i = 0; i < numElements; ++i) {
    error += fabs(h_C_cpu[i] - h_C_gpu[i]);
  }
  printf("Total difference between CPU and GPU results: %f\n", error);

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipEventDestroy(start_gpu);
  hipEventDestroy(stop_gpu);

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C_gpu);
  free(h_C_cpu);

  return 0;
}