#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 512

void checkCudaErrors(hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// Helper to calculate grid size (number of blocks)
inline int calculateGridSize(int N, int threadsPerBlock) {
  // Each block processes 2 * threadsPerBlock elements
  return (N + threadsPerBlock * 2 - 1) / (threadsPerBlock * 2);
}

__global__ void sumArray(const float *input, float *output, int N) {
  __shared__ float sharedData[THREADS_PER_BLOCK];

  // threadIdx.x is the thread index within the block
  // blockIdx.x is the block index within the grid
  // blockDim.x is the number of threads in the block
  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

  float sum = 0.0f;
  if (i < N) {
    sum += input[i];
    printf("Thread %d in Block %d processing element %d\n", tid, blockIdx.x, i);
    printf("Value: %f\n", input[i]);
    printf("Sum so far in Thread %d: %f\n", tid, sum);
  }
  if (i + blockDim.x < N) {
    sum += input[i + blockDim.x];
    printf("2nd Element Processing:\n");
    printf("Thread %d in Block %d processing element %d\n", tid, blockIdx.x,
           i + blockDim.x);
    printf("Value: %f\n", input[i + blockDim.x]);
    printf("Sum so far in Thread %d: %f\n", tid, sum);
  }
  sharedData[tid] = sum;

  __syncthreads();

  // Parallel reduction
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      sharedData[tid] += sharedData[tid + stride];
    }
    __syncthreads();
  }

  if (tid == 0)
    output[blockIdx.x] = sharedData[0];
}

int main() {
  int N = 1 << 5; // 1,048,576 elements
  const int threadsPerBlock = THREADS_PER_BLOCK;
  const int blocks = calculateGridSize(N, threadsPerBlock);

  float *h_array = new float[N];
  for (int i = 0; i < N; i++)
    h_array[i] = static_cast<float>(i);

  // CPU reference sum
  double cpuSum = 0.0;
  for (int i = 0; i < N; ++i)
    cpuSum += h_array[i];

  float *d_input, *d_output;
  checkCudaErrors(hipMalloc(&d_input, N * sizeof(float)));
  checkCudaErrors(hipMalloc(&d_output, blocks * sizeof(float)));

  checkCudaErrors(
      hipMemcpy(d_input, h_array, N * sizeof(float), hipMemcpyHostToDevice));

  sumArray<<<blocks, threadsPerBlock>>>(d_input, d_output, N);
  checkCudaErrors(hipDeviceSynchronize());

  // Retrieve partial sums
  float *partial = new float[blocks];
  checkCudaErrors(hipMemcpy(partial, d_output, blocks * sizeof(float),
                             hipMemcpyDeviceToHost));

  double gpuSum = 0.0;
  for (int i = 0; i < blocks; i++)
    gpuSum += partial[i];

  printf("Sum Computed on GPU : %.0f\n", gpuSum);
  printf("Sum Computed on CPU : %.0f\n", cpuSum);

  double epsilon = 1.0e-3;
  if (fabs(gpuSum - cpuSum) > epsilon)
    printf("Test Failed!\n");
  else
    printf("Test Passed!\n");

  delete[] h_array;
  delete[] partial;
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}
